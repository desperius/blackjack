// main.cu
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel(void)
{
    //
}

void launch(void)
{
   dim3 threads(2, 1);
   dim3 blocks(1, 1);

   kernel<<<blocks, threads>>>();

   int num_devices;
   int device;
   hipGetDeviceCount(&num_devices);
   printf("Device count: %d!\n", num_devices);

   if (num_devices > 1) 
   {
       int max_multiprocessors = 0;
       int max_device = 0;
       
       for (device = 0; device < num_devices; ++device) 
       {
           hipDeviceProp_t properties;
           hipGetDeviceProperties(&properties, device);
           printf("Device name: %s!\n", properties.name);

           if (max_multiprocessors < properties.multiProcessorCount) 
           {
               max_multiprocessors = properties.multiProcessorCount;
               max_device = device;
           }
       }

       hipSetDevice(max_device);
   }
}